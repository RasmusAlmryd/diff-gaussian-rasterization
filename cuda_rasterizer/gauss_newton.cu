#include "hip/hip_runtime.h"
#include "auxiliary.h"
#include "gauss_newton.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#define THREADS_PER_BLOCK 256


__global__
void createJ(){

}

__global__ void diagJTJ(const float* J, float* M_precon, uint32_t N, uint32_t M){
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= N*M) return;
    uint32_t x = idx % N;
    uint32_t y = idx / N;


    float j_val = J[x + y * N];

    float tmp = j_val * j_val;

    atomicAdd(&M_precon[x],tmp);
}

__global__ void next_z(float* M_precon, float* r_0, float* z, uint32_t N){
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= N) return;
    float denominator = M_precon[idx];
    if (denominator == 0){
        denominator = 0.000000001;
        // inv = 1.0;
    }
    float inv = 1.0 / denominator;
    float z_val = inv * r_0[idx];
    z[idx] = z_val;
}

// __global__ void rTr(float* r, float* R, uint32_t N){
//     float r_val = r[N];
//     atomicAdd(R, r_val*r_val);
// }

__global__ void dot(float *a, float *b, float *c, uint32_t N){
    __shared__ float temp[THREADS_PER_BLOCK];
    uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;

    if(idx >= N) return;

    temp[threadIdx.x] = a[idx] * b[idx];

    __syncthreads();

    if (threadIdx.x == 0)
    {
        float sum = 0;
        for (int i = 0; i < THREADS_PER_BLOCK; i++)
        {
            sum += temp[i];
        }
        atomicAdd(c, sum);
    }
}

// void dot(const float* a, const float* b, float* c, uint32_t N) {
//     shared float temp[THREADS_PER_BLOCK];
//     uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;

//     if (idx < N) {
//         temp[threadIdx.x] = a[idx] * b[idx];
//     } else {
//         temp[threadIdx.x] = 0.0f;
//     }

//     syncthreads();

//     // Perform reduction within the block
//     for (uint32_t stride = blockDim.x / 2; stride > 0; stride /= 2) {
//         if (threadIdx.x < stride) {
//             temp[threadIdx.x] += temp[threadIdx.x + stride];
//         }
//         syncthreads();
//     }

//     // Write the result of the block reduction to the output array
//     if (threadIdx.x == 0) {
//         atomicAdd(c, temp[0]);
//     }
// }

// template <unsigned int blockSize>
// __device__ void warpReduce(volatile int *sdata, unsigned int tid) {
//     if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
//     if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
//     if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
//     if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
//     if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
//     if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
// }


// template <unsigned int blockSize>
// __global__ void dot(int *g_idata, int *g_odata, unsigned int n) {
//     extern __shared__ int sdata[];
//     unsigned int tid = threadIdx.x;
//     unsigned int i = blockIdx.x*(blockSize*2) + tid;
//     unsigned int gridSize = blockSize*2*gridDim.x;
//     sdata[tid] = 0;
//     while (i < n) { sdata[tid] += g_idata[i] + g_idata[i+blockSize]; i += gridSize; }
//     __syncthreads();
//     if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
//     if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
//     if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
//     if (tid < 32) warpReduce(sdata, tid);
//     if (tid == 0) g_odata[blockIdx.x] = sdata[0];
// }

__global__
void gpu_copy(float* src, float* dest, const uint32_t N){
    uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx >= N) return;
    
    dest[idx] = src[idx];
}
__global__ 
void Scale(float* v, float* s, float* v_acc, uint32_t N){                // Used for calculating Ap in PCG
    uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx >= N) return;
    v_acc[idx] = *s * v[idx];
}

__global__
void sum_residuals(float* Ap, float* residual_dot_p, float* J, uint32_t N, uint32_t M){  // Sums over r(i) with scalar vector product r(i)^T * p
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= N*M) return;
    uint32_t x = idx % N;
    uint32_t y = idx / N;
    atomicAdd(&Ap[x], residual_dot_p[y] * J[x + y * N]);
}

__global__
void scalar_divide(float* numerator, float* denominator, float* quotient){
    *quotient = *numerator / *denominator;
}

__global__
void next_x(float* x, float* p, float* alpha, uint32_t N){
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= N) return;
    x[idx] = x[idx] + *alpha * p[idx];
}


__global__
void next_p(float* z, float* p, float* beta, uint32_t N){
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= N) return;
    p[idx] = z[idx] + *beta * p[idx];
}

__global__
void next_r(float* r, float* Ap, float* alpha, uint32_t N){
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= N) return;
    float new_val = r[idx] - *alpha * Ap[idx];
    r[idx] = new_val;
} 
void PCG(float* J, float gamma, float alpha){
    // Move PCG algo here from GaussNewton below.



}

void AP(float* J, float* p, float* Ap, uint32_t N, uint32_t M){
    float* residual_dot_p;
    hipMalloc(&residual_dot_p, M * sizeof(float));
    dot<<<(N+255)/256, 256>>>(J, p, residual_dot_p, N);  // r(i)^T * p
    sum_residuals<<<((N*M)+255)/256, 256>>>(Ap, residual_dot_p, J, N, M);    // Ap = r(i) * (r(i)^T * p)
    hipFree(residual_dot_p);
}



void GaussNewton::gaussNewtonUpdate(
    float* x,   // Is named delta in init.py : Check argument position.
    float* J,
    float* b,
    float gamma,
    float alpha,
    const bool* tiles_touched,
    const uint32_t N, // number of parameters
    const uint32_t M  // number of residuals
    ){

    printf("Address of x is %p\n", (void *)x);
    
    float* M_precon;
    hipMalloc(&M_precon, N * sizeof(float));
    dim3 threadsPerBlock(256); // thread block size: 256
    dim3 numBlocks(((N*M)+255)/256); 
    
    diagJTJ<<<numBlocks, threadsPerBlock>>>(J, M_precon, N, M);
    
    // r_0 = b
    // z_0 = M^-1*r_0
    // float* r= b;
    float* dev_r;
    hipMalloc(&dev_r, N * sizeof(float));
    gpu_copy<<<(N+255)/256, 256>>>(b, dev_r, N);

    float* dev_z;
    hipMalloc(&dev_z, N * sizeof(float));
    next_z<<<(N+255)/256, 256>>>(M_precon, dev_r, dev_z, N);
    
    float eps = 0.00001;
    float* h_R;
    float* h_R_prev;
    float* dev_R;
    float* dev_R_prev;
    float* dev_Ap;
    
    hipMalloc(&dev_R, sizeof(float));
    hipMalloc(&dev_R_prev, sizeof(float));
    hipMalloc(&dev_Ap, N * sizeof(float));
    
    h_R = (float *)malloc(sizeof(float));
    h_R_prev = (float *)malloc(sizeof(float));

    hipMemset(dev_R, 0, sizeof(float));
    hipMemset(dev_R_prev, 0, sizeof(float));
    
    dot<<<(N+255)/256, 256>>>(dev_r, dev_r, dev_R_prev, N); // R_prev = rTr
    
    
    float* dev_p;
    hipMalloc(&dev_p, N * sizeof(float));
    gpu_copy<<<(N+255)/256, 256>>>(dev_z, dev_p, N);
    
    
    float* dev_alpha;
    float* dev_denominator;
    float* dev_numerator;
    float* dev_beta;
    //    float* host_alpha;
    //    float* host_alpha_denominator;
    //    float* host_alpha_numerator;
    hipMalloc(&dev_alpha, sizeof(float));
    hipMalloc(&dev_denominator, sizeof(float));
    hipMalloc(&dev_numerator, sizeof(float));
    hipMalloc(&dev_beta, sizeof(float));

    
    float* h_float;
    h_float = (float *)malloc(sizeof(float));
    
    
    int k = 0; 
    const int MAX_ITERATIONS = 5;
    hipMemcpy(h_R_prev, dev_R_prev, sizeof(float), hipMemcpyDeviceToHost);
    
    while(k < MAX_ITERATIONS){
        hipMemset(dev_alpha, 0, sizeof(float));
        hipMemset(dev_denominator, 0, sizeof(float));
        hipMemset(dev_numerator, 0, sizeof(float));
        hipMemset(dev_beta, 0, sizeof(float));

        dot<<<(N+255)/256, 256>>>(dev_r, dev_z, dev_numerator, N); // r(k)^T * z(k)
        AP(J, dev_p, dev_Ap, N, M); 
        dot<<<(N+255)/256, 256>>>(dev_Ap, dev_p, dev_denominator, N); // p(k)^T * Ap(k) 
        scalar_divide<<<1, 1>>>(dev_numerator, dev_denominator, dev_alpha); // alpha = r(k)^T * z(k) / p(k)^T * Ap(k) 
        
        //Calculate next x
        // float temp_alpha = 0.2;
        hipMemcpy(h_float, dev_alpha, sizeof(float), hipMemcpyDeviceToHost);
        printf("dev_alpha: %f \n", *h_float);
        next_x<<<(N+255)/256, 256>>>(x, dev_p, dev_alpha, N);
        dot<<<(N+255)/256, 256>>>(dev_r, dev_z, dev_numerator, N); // r(k+1)^T * r(k+1), this is for the denomiator in the beta calculation
        next_r<<<(N+255)/256, 256>>>(dev_r, dev_Ap, dev_alpha, N);
        dot<<<(N+255)/256, 256>>>(dev_r, dev_r, dev_R, N); // R = r(k+1)^T * r(k+1)
        
        // Check if R/Rprev > 0.85 or R < eps
        hipMemcpy(h_R, dev_R, sizeof(float), hipMemcpyDeviceToHost);
        printf("R: %f, Rprev: %f \n", *h_R, *h_R_prev);
        if (*h_R/ *h_R_prev > 0.85 || *h_R < eps){
            break;
        }
        
        // if (*h_R < eps){
        //     break;
        // }

        // 
        *h_R_prev = *h_R;
        printf("R_prev: %f \n", *h_R_prev);
        next_z<<<(N+255)/256, 256>>>(M_precon, dev_r, dev_z, N);
        dot<<<(N+255)/256, 256>>>(dev_r, dev_z, dev_numerator, N); // r(k+1)^T * z(k+1)
        scalar_divide<<<1, 1>>>(dev_numerator, dev_denominator, dev_beta); // beta = r(k+1)^T * z(k+1) / r(k)^T * z(k)
        // break;

        // Calculate next p
        next_p<<<(N+255)/256, 256>>>(dev_z, dev_p, dev_beta, N);
        k++;

    }






    //Free memory
    free(h_R);
    free(h_R_prev);
    free(h_float);

    hipFree(dev_p);
    hipFree(dev_R);
    hipFree(dev_R_prev);
    hipFree(dev_Ap);
    hipFree(dev_alpha);
    hipFree(dev_denominator);
    hipFree(dev_numerator);
    hipFree(dev_beta);
    hipFree(dev_z);
    hipFree(M_precon);
    hipFree(dev_r);



}
